#include "hip/hip_runtime.h"
/*
 * Based on CSC materials from:
 * 
 * https://github.com/csc-training/openacc/tree/master/exercises/heat
 *
 */
#include <cmath>
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "pngwriter.h"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
#define BLOCK_SIZE_Z 16
/* Convert 2D index layout to unrolled 1D layout
 *
 * \param[in] i      Row index
 * \param[in] j      Column index
 * \param[in] width  The width of the area
 * 
 * \returns An index in the unrolled 1D array.
 */
__host__ __device__ int getIndex(const int i, const int j, const int k,  const int width, const int deep)
{
    return (i*width + j)*deep + k;
}

__global__ void heat_kernel(int nx, int ny, int nz, double* d_Un, double* d_Unp1, double aTimesDt, double dx2, double dy2, double dz2)
{
    // Going through the entire area
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i > 0 && i < nx-1)
    {
        int j = threadIdx.y + blockIdx.y*blockDim.y;
        if (j > 0 && j < ny-1)
        {
            int k = threadIdx.z + blockIdx.z*blockDim.z;
            if (k > 0 && k < nz-1) {
                const int index = getIndex(i, j, k, ny, nz);
                double uij = d_Un[index];
                double uim1jk = d_Un[getIndex(i-1, j, k, ny, nz)];
                double uijm1k = d_Un[getIndex(i, j-1, k, ny, nz)];
                double uijkm1 = d_Un[getIndex(i, j, k-1, ny, nz)];
                
                double uip1jk = d_Un[getIndex(i+1, j, k, ny, nz)];
                double uijp1k = d_Un[getIndex(i, j+1, k, ny, nz)];
                double uijkp1 = d_Un[getIndex(i, j, k+1, ny, nz)];
                
                // Explicit scheme
                d_Unp1[index] = uij + aTimesDt * ( (uim1jk - 2.0*uij + uip1jk)/dx2 + (uijm1k - 2.0*uij + uijp1k)/dy2+ (uijkm1 - 2.0*uij + uijkp1)/dz2);
            }
        }
    }
}


int main()
{
    const int nx = 200;   // Width of the area
    const int ny = 200;   // Height of the area
    const int nz = 200;   // Depth of the area

    double a;     // Diffusion constant
    std::cout << "Enter the diffusion constant (a): ";
    std::cin >> a;

    const double dx = 0.01;   // Horizontal grid spacing 
    const double dy = 0.01;   // Vertical grid spacing
    const double dz = 0.01;

    const double dx2 = dx*dx;
    const double dy2 = dy*dy;
    const double dz2 = dz*dz;

    const double dt = dx2 * dy2 * dz2/ (2.0 * a * (dx2 + dy2 + dz2)); // Largest stable time step
    const int numSteps = 500000;                                       // Number of time steps
    const int outputEvery = 1000;                                    // How frequently to write output image

    int numElements = nx*ny*nz;

    // Allocate two sets of data for current and next timesteps
    double* h_Un   = (double*)calloc(numElements, sizeof(double));
    double* h_Unp1 = (double*)calloc(numElements, sizeof(double));

    double* d_Un;
    double* d_Unp1;

    hipMalloc((void**)&d_Un, numElements*sizeof(double));
    hipMalloc((void**)&d_Unp1, numElements*sizeof(double));

    
    // Initializing the data with a pattern of disk of radius of 1/6 of the width
    double object_x;
    std::cout << "Enter the width of the object ";
    std::cin >> object_x;
    double object_y;
    std::cout << "Enter the height of the object ";
    std::cin >> object_y;
    double object_z;
    std::cout << "Enter the deep of the object ";
    std::cin >> object_z;
    
    double center_x = nx/2;
    double center_y = ny/2;
    double center_z = nz/2;

    for (int i = 0; i < nx; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            for (int k = 0; k < nz; ++k) {
                int index = getIndex(i, j, k, ny, nz);
                // Distance of point i, j from the origin
                if ((abs(i - center_x) <= object_x / 2) && (abs(j - center_y) <= object_y / 2) && (abs(k - center_z) <= object_z / 2)) {
                    h_Un[index] = 10.0;
                } else {
                    h_Un[index] = 5.0;
                }
            }
        }
    }

    // Fill in the data on the next step to ensure that the boundaries are identical.
    memcpy(h_Unp1, h_Un, numElements*sizeof(double));

    hipMemcpy(d_Un, h_Un, numElements*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Unp1, d_Un, numElements*sizeof(double), hipMemcpyDeviceToDevice);

    // Timing
    clock_t start = clock();

    dim3 numBlocks(nx/BLOCK_SIZE_X + 1, ny/BLOCK_SIZE_Y + 1, nz/BLOCK_SIZE_Z + 1);
    dim3 threadsPerBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);

    // Main loop
    for (int n = 0; n <= numSteps; n++)
    {
        heat_kernel<<<numBlocks, threadsPerBlock>>>(nx, ny, nz, d_Un, d_Unp1, a*dt, dx2, dy2, dz2);
        // Write the output if needed
        if (n % outputEvery == 0)
        {
            hipMemcpy(h_Un, d_Un, numElements*sizeof(double), hipMemcpyDeviceToHost);

            int sum_temp_in = 0;
            int number_in = 0;
            int sum_temp_out = 0;
            int number_out = 0;
            double res = -10000000000;
            for (int i = 0; i < nx; i++)
            {
                for (int j = 0; j < ny; j++)
                {
                    for (int k = 0; k < nz; ++k) {
                        int index = getIndex(i, j,k, ny, nz);
                        // Distance of point i, j from the origin
                        if ((abs(i - center_x) <= object_x / 2) && (abs(j - center_y) <= object_y / 2) && (abs(k - center_z) <= object_z / 2)) {
                            sum_temp_in += h_Un[index];
                            ++number_in;
                        } else {
                            sum_temp_out += h_Un[index];
                            ++number_out;
                        }

                        double uij = h_Un[index];
                        double uim1jk = h_Un[getIndex(i-1, j, k, ny, nz)];
                        double uijm1k = h_Un[getIndex(i, j-1, k, ny, nz)];
                        double uijkm1 = h_Un[getIndex(i, j, k-1, ny, nz)];
                        
                        double uip1jk = h_Un[getIndex(i+1, j, k, ny, nz)];
                        double uijp1k = h_Un[getIndex(i, j+1, k, ny, nz)];
                        double uijkp1 = h_Un[getIndex(i, j, k+1, ny, nz)];
                        
                        // Explicit scheme
                        res = std::max(a*dt * ( (uim1jk - 2.0*uij + uip1jk)/dx2 + (uijm1k - 2.0*uij + uijp1k)/dy2 + (uijkm1 - 2.0*uij + uijkp1)/dz2), res);
                            }
                }
            }
            std::cout << res << std::endl;
            std::cout << "Mean temperature in the start zone" << sum_temp_in << " " << number_in << ", out: " << sum_temp_out << " " <<  number_out << std::endl;
            //save_stats(h_Un, nx, ny, nz, object_x, object_y, object_z, filename, 'c');
        }
        // Swapping the pointers for the next timestep
        std::swap(d_Un, d_Unp1);
    }

    // Timing
    clock_t finish = clock();
    printf("It took %f seconds\n", (double)(finish - start) / CLOCKS_PER_SEC);

    // Release the memory
    hipFree(d_Un);
    hipFree(d_Unp1);
    free(h_Un);
    free(h_Unp1);
    
    return 0;
}